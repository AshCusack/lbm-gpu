#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>

#include "lbm_solver_gpu.h"
#include "lbm_model.h"
#include "utils_gpu.h"
#include "utils.h"
#include "cell_computation_gpu.cuh"

__constant__ float tau_d, wall_velocity_d[D_LBM];
__constant__ int xlength_d, num_cells_d;


/**
 * Computes the density from the particle distribution functions stored at currentCell.
 * currentCell thus denotes the address of the first particle distribution function of the
 * respective cell. The result is stored in density.
 */
__device__ void ComputeDensityGpu(float *current_cell, float *density){
    int i; *density=0;
    //TODO:get rid of this loop
    for(i=0;i<Q_LBM;i++)
        *density+=current_cell[i];
    /* TODO:Density should be close to a unit (ρ~1) */
}


/**
 * Computes the velocity within currentCell and stores the result in velocity
 */
__device__ void ComputeVelocityGpu(float *current_cell, float *density, float *velocity){
    int i;
    velocity[0]=0;
    velocity[1]=0;
    velocity[2]=0;

    //TODO:get rid of this loop
    for(i=0;i<Q_LBM;i++){
        velocity[0]+=current_cell[i]*LATTICE_VELOCITIES_D[i][0];
        velocity[1]+=current_cell[i]*LATTICE_VELOCITIES_D[i][1];
        velocity[2]+=current_cell[i]*LATTICE_VELOCITIES_D[i][2];
    }

    velocity[0]/=*density;
    velocity[1]/=*density;
    velocity[2]/=*density;
}

/**
 * Computes the equilibrium distributions for all particle distribution functions of one
 * cell from density and velocity and stores the results in feq.
 */
__device__ void ComputeFeqGpu(float *density, float *velocity, float *feq){
    int i; float s1, s2, s3;
    //TODO:get rid of this loop
    for(i=0;i<Q_LBM;i++){
        s1 = LATTICE_VELOCITIES_D[i][0]*velocity[0]+LATTICE_VELOCITIES_D[i][1]*velocity[1]+
        		LATTICE_VELOCITIES_D[i][2]*velocity[2];
        s2 = s1*s1;
        s3 = velocity[0]*velocity[0]+velocity[1]*velocity[1]+velocity[2]*velocity[2];

        feq[i]=LATTICE_WEIGHTS_D[i]*(*density)*(1+s1*C_S_POW2_INV+s2*C_S_POW4_INV/2.0-s3*C_S_POW2_INV/2.0);
        /* TODO:Probability distribution function can not be less than 0 */
    }
}


/**
 * Computes the post-collision distribution functions according to the BGK update rule and
 * stores the results again at the same position.
 */
__device__ void ComputePostCollisionDistributionsGpu(float *current_cell, float *feq){
    int i;
    //TODO:get rid of this loop
    for(i=0;i<Q_LBM;i++){
        current_cell[i]=current_cell[i]-(current_cell[i]-feq[i])/tau_d;

        /* TODO:Probability distribution function can not be less than 0 */
    }
}

// TODO: rename in inv
__device__ int inv2(int i){
    return (Q_LBM-1)-i;
}


/**
 * Performs the actual collision computation
 */
__global__ void DoColision(float *collide_field_d){
	float density, velocity[D_LBM], feq[Q_LBM], *current_cell_s;
	__shared__ float collide_field_s[BLOCK_SIZE*BLOCK_SIZE*BLOCK_SIZE*Q_LBM];
	//TODO:can be optimized using BLOCK_SIZE constant
	int x = 1+threadIdx.x+blockIdx.x*blockDim.x;
	int y = 1+threadIdx.y+blockIdx.y*blockDim.y;
	int z = 1+threadIdx.z+blockIdx.z*blockDim.z;
	int idx_block = threadIdx.x+threadIdx.y*blockDim.x+threadIdx.z*blockDim.x*blockDim.y;
	int step = xlength_d+2, i;
	int idx_domain = x+y*step+z*step*step;

	//check that indices are within the bounds since there could be more threads than needed
	if (x<(step-1) && y<(step-1) && z<(step-1)){
		//copy current cell values into shared memory
		for(i=0;i<Q_LBM;i++)
			collide_field_s[Q_LBM*idx_block+i]=collide_field_d[Q_LBM*idx_domain+i];

		current_cell_s = &collide_field_s[Q_LBM*idx_block];
		//perform computation
		ComputeDensityGpu(current_cell_s,&density);
		ComputeVelocityGpu(current_cell_s,&density,velocity);
		ComputeFeqGpu(&density,velocity,feq);
		ComputePostCollisionDistributionsGpu(current_cell_s,feq);

		//copy data back
		for(i=0;i<Q_LBM;i++)
			collide_field_d[Q_LBM*idx_domain+i]=collide_field_s[Q_LBM*idx_block+i];
	}
}


/**
 * Performs the actual streaming computation
 */
__global__ void DoStreaming(float *stream_field_d, float *collide_field_d){
	//	__syncthreads(); to use after reading data into shared memory
	int x = 1+threadIdx.x+blockIdx.x*blockDim.x;
	int y = 1+threadIdx.y+blockIdx.y*blockDim.y;
	int z = 1+threadIdx.z+blockIdx.z*blockDim.z;
	int step = xlength_d+2, idx = x+y*step+z*step*step, nx, ny, nz, i;

	//check that indices are within the bounds since there could be more threads than needed
	if (x<(step-1) && y<(step-1) && z<(step-1)){
		for(i=0;i<Q_LBM;i++){
			nx=x-LATTICE_VELOCITIES_D[i][0];
			ny=y-LATTICE_VELOCITIES_D[i][1];
			nz=z-LATTICE_VELOCITIES_D[i][2];

			stream_field_d[Q_LBM*idx+i]=collide_field_d[Q_LBM*(nx+ny*step+nz*step*step)+i];
		}
	}
}

__global__ void TreatBoundary(float *collide_field_d, int* flag_field_d){
	int x = threadIdx.x+blockIdx.x*blockDim.x;
	int y = threadIdx.y+blockIdx.y*blockDim.y;
	int z = threadIdx.z+blockIdx.z*blockDim.z;
    int nx,ny,nz,i,step=xlength_d+2;
    float density,dot_prod;

    if (x<step && y<step && z<step){
		if(flag_field_d[x+y*step+z*step*step]!=FLUID){
			for(i=0;i<Q_LBM;i++){
				nx=x+LATTICE_VELOCITIES_D[i][0];
				ny=y+LATTICE_VELOCITIES_D[i][1];
				nz=z+LATTICE_VELOCITIES_D[i][2];

				/* We don't need the values outside of our extended domain */
				if(0<nx && nx<step-1 && 0<ny && ny<step-1 && 0<nz && nz<step-1){
					if (flag_field_d[x+y*step+z*step*step]==MOVING_WALL){
						/* Compute density in the neighbour cell */
						ComputeDensityGpu(&collide_field_d[Q_LBM*(nx+ny*step+nz*step*step)],&density);
						/* Compute dot product */
						dot_prod=LATTICE_VELOCITIES_D[i][0]*wall_velocity_d[0]+
								LATTICE_VELOCITIES_D[i][1]*wall_velocity_d[1]+
								LATTICE_VELOCITIES_D[i][2]*wall_velocity_d[2];
						/* Assign the boudary cell value */
						collide_field_d[Q_LBM*(x+y*step+z*step*step)+i]=
								collide_field_d[Q_LBM*(nx+ny*step+nz*step*step)+inv2(i)]+
								2*LATTICE_WEIGHTS_D[i]*density*C_S_POW2_INV*dot_prod;
					}else if(flag_field_d[x+y*step+z*step*step]==NO_SLIP){
						collide_field_d[Q_LBM*(x+y*step+z*step*step)+i]=
								collide_field_d[Q_LBM*(nx+ny*step+nz*step*step)+inv2(i)];
					}
				}
			}
		}
    }
}


void DoCollisionGpu(float *collide_field, int *flag_field, float tau, int xlength){
	float *collide_field_d=NULL;
	int num_cells = pow(xlength+2, D_LBM);
	size_t collide_field_size = Q_LBM*num_cells*sizeof(float);

	//initialize working data
	cudaErrorCheck(hipMalloc(&collide_field_d, collide_field_size));
	cudaErrorCheck(hipMemcpy(collide_field_d, collide_field, collide_field_size, hipMemcpyHostToDevice));

	//initialize constant data
	cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(tau_d), &tau, sizeof(float), 0, hipMemcpyHostToDevice));
	cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(xlength_d), &xlength, sizeof(int), 0, hipMemcpyHostToDevice));

	//define grid structure
	//NOTE:redundant threads for boundary cells are not accounted for
	dim3 block(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid((xlength+block.x-1)/block.x, (xlength+block.y-1)/block.y, (xlength+block.z-1)/block.z);

	//perform collision
	DoColision<<<grid,block>>>(collide_field_d);
	cudaErrorCheck(hipPeekAtLastError());

	//copy data back to host
	cudaErrorCheck(hipMemcpy(collide_field, collide_field_d, collide_field_size, hipMemcpyDeviceToHost));

	//free device memory
	cudaErrorCheck(hipFree(collide_field_d));
}


void TreatBoundaryGpu(float *collide_field, int *flag_field, float *wall_velocity, int xlength){
	float *collide_field_d=NULL, data[3];
	int *flag_field_d=NULL, num_cells;
	size_t size;

	for(int i=0;i<D_LBM;i++)
		data[i]=wall_velocity[i];

	hipMemcpyToSymbol(HIP_SYMBOL(wall_velocity_d), data, sizeof(data), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(xlength_d), &xlength, sizeof(int), 0, hipMemcpyHostToDevice);

	num_cells = (xlength+2)*(xlength+2)*(xlength+2);
	size = Q_LBM*num_cells*sizeof(float);

	hipMalloc(&collide_field_d, size);
	hipMalloc(&flag_field_d, num_cells*sizeof(int));
	hipMemcpy(collide_field_d, collide_field, size, hipMemcpyHostToDevice);
	hipMemcpy(flag_field_d, flag_field, num_cells*sizeof(int), hipMemcpyHostToDevice);

	//define grid structure
	//NOTE:redundant threads for boundary cells are not accounted for
	dim3 block(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid((xlength+2+block.x-1)/block.x, (xlength+2+block.y-1)/block.y, (xlength+2+block.z-1)/block.z);

	TreatBoundary<<<grid,block>>>(collide_field_d, flag_field_d);
	cudaErrorCheck(hipPeekAtLastError());

	hipMemcpy(collide_field, collide_field_d, size, hipMemcpyDeviceToHost);
	hipFree(collide_field_d);
	hipFree(flag_field_d);
}


void DoStreamingGpu(float *collide_field, float *stream_field, int *flag_field, int xlength){
	float *collide_field_d=NULL, *stream_field_d=NULL;
	int num_cells = pow(xlength+2, D_LBM);
	size_t computational_field_size = Q_LBM*num_cells*sizeof(float);

	//initialize working data
	cudaErrorCheck(hipMalloc(&collide_field_d, computational_field_size));
	cudaErrorCheck(hipMemcpy(collide_field_d, collide_field, computational_field_size, hipMemcpyHostToDevice));
	cudaErrorCheck(hipMalloc(&stream_field_d, computational_field_size));
	cudaErrorCheck(hipMemcpy(stream_field_d, stream_field, computational_field_size, hipMemcpyHostToDevice));

	//initialize constant data
	cudaErrorCheck(hipMemcpyToSymbol(HIP_SYMBOL(xlength_d), &xlength, sizeof(int), 0, hipMemcpyHostToDevice));

	//define grid structure
	//NOTE:redundant threads for boundary cells are not accounted for
	dim3 block(BLOCK_SIZE, BLOCK_SIZE, BLOCK_SIZE);
	dim3 grid((xlength+block.x-1)/block.x, (xlength+block.y-1)/block.y, (xlength+block.z-1)/block.z);

	//perform streaming
	DoStreaming<<<grid,block>>>(stream_field_d, collide_field_d);
	cudaErrorCheck(hipPeekAtLastError());

	//copy data back to host
	cudaErrorCheck(hipMemcpy(stream_field, stream_field_d, computational_field_size, hipMemcpyDeviceToHost));

	//free device memory
	cudaErrorCheck(hipFree(collide_field_d));
	cudaErrorCheck(hipFree(stream_field_d));
}
