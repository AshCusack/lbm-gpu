#include "hip/hip_runtime.h"
#include "initialization_gpu.h"
#include "lbm_model.h"
#include "utils_gpu.h"

void InitialiseDeviceFields(float *collide_field, float *stream_field,int *flag_field, int xlength, float **collide_field_d, float **stream_field_d,int **flag_field_d){
	int num_cells = pow(xlength+2, D_LBM);
	size_t computational_field_size = Q_LBM*num_cells*sizeof(float);
	size_t flag_field_size = num_cells*sizeof(int);

	cudaErrorCheck(hipMalloc(collide_field_d, computational_field_size));
	cudaErrorCheck(hipMemcpy(*collide_field_d, collide_field, computational_field_size, hipMemcpyHostToDevice));
	cudaErrorCheck(hipMalloc(stream_field_d, computational_field_size));
	cudaErrorCheck(hipMemcpy(*stream_field_d, stream_field, computational_field_size, hipMemcpyHostToDevice));
	cudaErrorCheck(hipMalloc(flag_field_d, flag_field_size));
	cudaErrorCheck(hipMemcpy(*flag_field_d, flag_field, flag_field_size, hipMemcpyHostToDevice));
}


void FreeDeviceFields(float **collide_field_d, float **stream_field_d,int **flag_field_d){
	cudaErrorCheck(hipFree(*collide_field_d));
	cudaErrorCheck(hipFree(*stream_field_d));
	cudaErrorCheck(hipFree(*flag_field_d));
}
