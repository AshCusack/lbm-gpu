#include <stdio.h>

#include "utils_gpu.h"


int HasCudaGpu(){
	int devices = 0;
	hipError_t err = hipGetDeviceCount(&devices);
	devices = (devices > 0 && err == hipSuccess) ? 1 : 0;
	return devices;
}
